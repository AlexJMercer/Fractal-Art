#include "hip/hip_runtime.h"
#include "fractals.hpp"

#include <SFML/Graphics.hpp>

#include <GL/gl.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_complex.h>

#include <stdio.h>

__global__ void calculateFractalKernel(unsigned char *pixelData, uint32_t WIDTH, uint32_t HEIGHT, sf::Color *paletteData, size_t paletteSize,
                                       double min_Re, double max_Re, double min_Im, double max_Im, uint32_t MAX_ITERATIONS)

{
    uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < WIDTH && y < HEIGHT)
    {
        double real = min_Re + x * (max_Re - min_Re) / WIDTH;
        double imag = min_Im + y * (max_Im - min_Im) / HEIGHT;

        hipDoubleComplex constant = make_hipDoubleComplex(real, imag);
        hipDoubleComplex z = make_hipDoubleComplex(0, 0);

        uint16_t n = 0;
        for (; n < MAX_ITERATIONS; ++n)
        {
            if (hipCabs(z) > 2.0)
                break;
            z = hipCadd(hipCmul(z, z), constant);
        }

        sf::Color pixelColor = (n == MAX_ITERATIONS) ? sf::Color::Black : paletteData[n % paletteSize];
        pixelData[4 * (y * WIDTH + x) + 0] = pixelColor.r;
        pixelData[4 * (y * WIDTH + x) + 1] = pixelColor.g;
        pixelData[4 * (y * WIDTH + x) + 2] = pixelColor.b;
    }
}


void Fractals::calculateCUDA()
{
    // Create sf window and display a red image
    sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Fractals");
    window.setFramerateLimit(60);

    // Create an image
    const uint32_t imageSize = WIDTH * HEIGHT;

    // Create a texture and sprite
    sf::Texture texture;
    texture.create(WIDTH, HEIGHT);
    
    
    // Get texture ID
    GLuint textureID = texture.getNativeHandle();
    
    // Register texture with CUDA
    hipGraphicsResource *cudaResource;
    cudaGraphicsRegisterImage(&cudaResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
    
    
    // Set up GRID & BLOCK Dimensions
    dim3 blockDIM(32, 32);
    dim3 gridDIM( (WIDTH + blockDIM.x - 1) / blockDIM.x, 
                  (HEIGHT + blockDIM.y - 1) / blockDIM.y );
    
    // Setup and Allocate Memory for Color Palette
    sf::Color *paletteData;
    hipMalloc(&paletteData, sizeof(sf::Color) * palette.colors.size());
    hipMemcpy(paletteData, palette.colors.data(), sizeof(sf::Color) * palette.colors.size(), hipMemcpyHostToDevice);
    
    // Window Loop
    sf::Sprite sprite(texture);

    while (window.isOpen())
    {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();
        }

        hipGraphicsMapResources(1, &cudaResource);
        hipArray* hipArray;
        hipGraphicsSubResourceGetMappedArray(&hipArray, cudaResource, 0, 0);


        unsigned char* pixelData;
        hipMemcpyFromArray(&pixelData, hipArray, 0, 0, imageSize, hipMemcpyDeviceToHost);

        // Launch Kernel
        calculateFractalKernel<<<gridDIM, blockDIM>>>(pixelData, WIDTH, HEIGHT, paletteData, palette.colors.size(), min_Re, max_Re, min_Im, max_Im, MAX_ITERATIONS);

        hipGraphicsUnmapResources(1, &cudaResource);

        window.clear();
        window.draw(sprite);
        window.display();
    }

    // Unregister texture with CUDA
    hipFree(paletteData);
    hipGraphicsUnregisterResource(cudaResource);
}